
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 10
#define THREADS 10
#define BLOCKS 1

__global__ void reversing(double* arr, double* res, int size){
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  res[size - 1 - index] = arr[index];
}

__global__ void transpose(double* mat, double* res, int ncol){
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  
  // global index associated to matrix
  int r = index / ncol;
  int c = index % ncol;

  // transposition
  res[c * ncol + r] = mat[index];
}

__global__ void mm_mult(double* A, double* B, double* C, int nr_A, int nc_A, int nc_B){

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  
  // global index associated to matrix
  int r = index / nc_B;
  int c = index % nc_B;
  int k;
  double tmp = 0.;

  for(k = 0; k < nc_A; k++)
    tmp += A[k + r * nc_A] * B[c + k * nc_A];

  C[index] = tmp;
}

int main(int argc, char** argv){

  int i, j, i_tmp;
  double* arr = (double*)malloc(N * sizeof(double));
  double *dev_arr, *res;

  double* mat = (double*)malloc(N * N * sizeof(double));
  double *dev_mat, *res_mat;
  double *identity = (double*)malloc(N * N * sizeof(double));
  double *dev_id, *dev_prod;

  if(THREADS * BLOCKS != N){
    printf("\n\tTHREADS * BLOCK must be equal to N\n\tExit\n");
    return 0;
  }
  
  hipMalloc(&dev_arr, N * sizeof(double));
  hipMalloc(&res, N * sizeof(double));

  for(i = 0; i < N; i++)
    arr[i] = (double) i;

  hipMemcpy(dev_arr, arr, N * sizeof(double), hipMemcpyHostToDevice);

  reversing<<<THREADS, BLOCKS>>>(dev_arr, res, N);

  hipMemcpy(arr, res, N * sizeof(double), hipMemcpyDeviceToHost);

  for(i = 0; i < N; i++)
    printf("\t%lg", arr[i]);
  printf("\n");

  hipFree(dev_arr);
  hipFree(res);
  free(arr);

  printf("\n\tTRANSPOSE\n\n");

  for(i = 0; i < N; i++){
    i_tmp = i * N;
    for(j = 0; j < N; j++){
      mat[j + i_tmp] = j + i_tmp;
      if(i != j)
	identity[i_tmp + j] = 0.;
      else
	identity[i_tmp + j] = 1.;
    }
  }

  for(i = 0; i < N; i++){
    i_tmp = i * N;

    for(j = 0; j < N; j++)
      printf("\t%lg", mat[i_tmp + j]);

    printf("\n");
}

  hipMalloc(&dev_mat, N * N * sizeof(double));
  hipMalloc(&res_mat, N * N * sizeof(double));
  hipMalloc(&dev_id,  N * N * sizeof(double));
  hipMalloc(&dev_prod,N * N * sizeof(double));

  hipMemcpy(dev_mat, mat, N * N * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_id, identity, N * N * sizeof(double), hipMemcpyHostToDevice);

  transpose<<<THREADS, THREADS>>>(dev_mat, res_mat, N);
  mm_mult<<<THREADS, THREADS>>>(res_mat, dev_id, dev_prod, N, N, N);

  hipMemcpy(mat, dev_prod, N * N * sizeof(double), hipMemcpyDeviceToHost);
  printf("\n\n");
  for(i = 0; i < N; i++){

    i_tmp = i * N;

    for(j = 0; j < N; j++)
      printf("\t%lg", mat[i_tmp + j]);
    printf("\n");
  }
  printf("\n");

  return 0;
}
